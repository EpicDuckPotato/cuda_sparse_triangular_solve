#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>

#include "cudaSolver.h"

CudaSolver::CudaSolver(int *row_idx, int *col_idx, double *vals, int rows, int nonzeros, double *b, bool spd) {
  hipsparseCreate(&cs_handle);

  hipMalloc(&device_row_ptr, sizeof(int)*(m + 1));
  hipMalloc(&device_col_idx, sizeof(int)*nnz);
  hipMalloc(&device_vals, sizeof(double)*nnz);

  int *device_row_idx;
  hipMalloc(&device_row_idx, sizeof(int)*nnz);
  hipMemcpy(device_row_idx, row_idx, sizeof(int)*nnz, hipMemcpyHostToDevice);
  hipMemcpy(device_col_idx, col_idx, sizeof(int)*nnz, hipMemcpyHostToDevice);
  hipMemcpy(device_vals, vals, sizeof(double)*nnz, hipMemcpyHostToDevice);
  hipMemcpy(device_b, b, sizeof(double)*m, hipMemcpyHostToDevice);

  // Convert COO to CSR
  hipsparseXcoo2csr(cs_handle, device_row_idx, nnz, m, device_row_ptr,
                   HIPSPARSE_INDEX_BASE_ZERO);

  hipFree(device_row_idx);

  use_cholesky = spd;
  m = m;
  nnz = nnz;

  lpop = false;
}

CudaSolver::~CudaSolver() {
  hipsparseDestroy(cs_handle);
  hipFree(device_row_ptr);
  hipFree(device_col_idx);
  hipFree(device_vals);
  hipFree(device_b);

  if (lpop) {
    hipFree(L_vals);
  }
}

void CudaSolver::factor() {
  // For the sake of getting things working, we'll just handle the spd case for now.
  // Later on, we can case on spd and choose Cholesky vs LU accordingly
  if (!use_cholesky) {
    return;
  }

  // Boilerplate
  csric02Info_t info;
  hipsparseCreateCsric02Info(&info);

  hipsparseMatDescr_t descr;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
  hipsparseSetMatFillMode(descr, HIPSPARSE_FILL_MODE_LOWER);
  hipsparseSetMatDiagType(descr, HIPSPARSE_DIAG_TYPE_NON_UNIT);

  int bufferSize;
  hipsparseDcsric02_bufferSize(cs_handle, m, nnz, descr, device_vals, device_row_ptr,
                              device_col_idx, info, &bufferSize);

  void *pBuffer;
  hipMalloc(&pBuffer, bufferSize);

  // Analyze
  hipsparseDcsric02_analysis(cs_handle, m, nnz, descr, device_vals,
                            device_row_ptr, device_col_idx, info,
                            HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);

  // Factor, and put the Cholesky factor into L_vals
  hipsparseDcsric02(cs_handle, m, nnz, descr, L_vals,
                   device_row_ptr, device_col_idx, info,
                   HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);

  hipFree(pBuffer);

  hipsparseDestroyMatDescr(descr);

  hipsparseDestroyCsric02Info(info);
  lpop = true;
}

void CudaSolver::solve(double *x) {
  // For the sake of getting things working, we'll just handle the spd case for now.
  // Later on, we can case on spd and choose Cholesky vs LU accordingly
  if (!use_cholesky) {
    return;
  }

  int *levelInd;
  int *levelPtr;
  int *chainPtr;
  int *rRoot;
  int *wRoot;
  int *cRoot;

  // Analysis phase
  /*
  dim3 blockDim(16, 16);
  kernelFindRoots<<<gridDim, blockDim>>>();
  */
}
