#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include "cudaSolver.h"
#include <malloc.h>

struct GlobalConstants {
  int *row_ptr;
  int *col_idx;
  int m;
  int nnz;
};

__constant__ GlobalConstants cuConstSolverParams;

/*
 * kernelFindRootsP1: parallelizes over rows of the dependency
 * graph and indicates roots
 * ARGUMENTS
 * roots: roots[i] is populated with 1 if row i is a root, and zero otherwise
 * depGraph: value array for the dependency graph
 */
__global__ void kernelFindRootsP1(int *roots, char *depGraph) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < cuConstSolverParams.m) {
    int rowStart = cuConstSolverParams.row_ptr[row];

    // There's a - 1 because the last element of the row
    // is the diagonal element, which isn't a dependency
    // for solving this row
    int rowEnd = cuConstSolverParams.row_ptr[row + 1] - 1;

    roots[row] = 1;
    for (int i = rowStart; i < rowEnd; ++i) {
      if (depGraph[i]) {
        // Dependency exists
        roots[row] = 0;
        break;
      }
    }
  }
}

/*
 * kernelFindRootsP2: should be called after kernelFindRootsP1
 * ARGUMENTS
 * wRoot: populated with rows of roots
 * nRoots: populated with number of roots
 * rootScan: inclusive scan of the roots array from kernelFindRootsP1
 */
__global__ void kernelFindRootsP2(int *wRoot, int *nRoots, int *rootScan) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < cuConstSolverParams.m &&
      ((row == 0 && rootScan[row] == 1) ||
       (row > 0 && rootScan[row] == rootScan[row - 1] + 1))) {
    wRoot[rootScan[row] - 1] = row;
  }
  *nRoots = rootScan[cuConstSolverParams.m - 1];
}

/*
 * kernelFindRootsInCandidatesP1: parallelizes over rows of the dependency
 * graph and indicates roots, only looking at rows given by cRoots
 * ARGUMENTS
 * roots: roots[i] is populated with 1 if row i is a root, and zero otherwise
 * cRoots: set of rows that could be roots
 * nCand: number of candidates
 * depGraph: value array for the dependency graph
 */
__global__ void kernelFindRootsInCandidatesP1(int *roots, int *cRoots, int *nCand, char *depGraph) {
  int candIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if (candIdx < *nCand) {
    int row = cRoots[candIdx];
    int rowStart = cuConstSolverParams.row_ptr[row];

    // There's a - 1 because the last element of the row
    // is the diagonal element, which isn't a dependency
    // for solving this row
    int rowEnd = cuConstSolverParams.row_ptr[cRoots[row] + 1] - 1;

    roots[row] = 1;
    for (int i = rowStart; i < rowEnd; ++i) {
      if (depGraph[i]) {
        // Dependency exists
        roots[row] = 0;
        break;
      }
    }
  }
}

/*
 * kernelAnalyze: populates levelInd, levelPtr, chainPtr, and cRoots.
 * chainPtr determines the properties and number of kernels to be launched in the solve phase.
 * ARGUMENTS
 * roots: populated with rows of roots
 * nRoots: number of roots
 * cRoots: set of rows that could be roots
 * nCand: number of candidates
 * levelInd: list of sorted rows belonging to every level
 * levelPtr: list of ending index (in levelInd) of each level
 * chainPtr: list of ending index (in levelPtr) of each chain
 * (Note: Naumov has levelPtr & chainPtr be the list of starting indices of each level/chain
 *   + an extra element to indicate the end of the last level/chain)
 * levelIndSize: size of levelInd
 * levelPtrSize: size of levelPtr
 * chainPtrSize: size of chainPtr
 * depGraph: value array for the dependency graph
 */
__global__ void kernelAnalyze(int *roots, int *nRoots, int *cRoots, int *nCand, int *levelInd, int *levelPtr, int *chainPtr, int *levelIndSize, int *levelPtrSize, int *chainPtrSize, char *depGraph) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < *nRoots) {
    // TODO: Not sure if this indexing is correct
    int root = roots[idx];
    int colStart = cuConstSolverParams.col_idx[root];
    int colEnd = cuConstSolverParams.col_idx[root + 1];

    for (int i = colStart; i < colEnd; ++i) {
      if (depGraph[i]) {
        // Dependency exists, set to 0 and add to cRoots
        depGraph[i] = 0;
        cRoots[*nCand] = i;
        *nCand += 1;
      }
    }

    // TODO: Populate levelInd, is this sorted? when to increase levelIndSize?
    levelInd[*levelIndSize + idx] = root;

    // Populate levelPtr, only do this once
    if (idx == *nRoots - 1) {
      levelPtr[*levelPtrSize] = *levelIndSize + *nRoots - 1;
      *levelPtrSize += 1;
    }

    // TODO: how to populate chainPtr? how to determine size of chain?
  }
}

CudaSolver::CudaSolver(int *row_idx, int *col_idx, double *vals, int m, int nnz, double *b, bool spd, bool is_lt) : m(m), nnz(nnz), spd(spd), is_lt(is_lt) {
  hipsparseCreate(&cs_handle);

  hipMalloc(&device_row_ptr, sizeof(int)*(m + 1));
  hipMalloc(&device_col_idx, sizeof(int)*nnz);
  hipMalloc(&device_vals, sizeof(double)*nnz);

  int *device_row_idx;
  hipMalloc(&device_row_idx, sizeof(int)*nnz);
  hipMemcpy(device_row_idx, row_idx, sizeof(int)*nnz, hipMemcpyHostToDevice);
  hipMemcpy(device_col_idx, col_idx, sizeof(int)*nnz, hipMemcpyHostToDevice);
  hipMemcpy(device_vals, vals, sizeof(double)*nnz, hipMemcpyHostToDevice);
  hipMemcpy(device_b, b, sizeof(double)*m, hipMemcpyHostToDevice);

  // Convert COO to CSR
  hipsparseXcoo2csr(cs_handle, device_row_idx, nnz, m, device_row_ptr,
                   HIPSPARSE_INDEX_BASE_ZERO);

  hipFree(device_row_idx);

  if (is_lt) {
    hipMalloc(&L_vals, sizeof(double)*nnz);
    hipMemcpy(L_vals, vals, sizeof(double)*nnz, hipMemcpyHostToDevice);
  }
  lpop = is_lt;
}

CudaSolver::~CudaSolver() {
  hipsparseDestroy(cs_handle);
  hipFree(device_row_ptr);
  hipFree(device_col_idx);
  hipFree(device_vals);
  hipFree(device_b);

  if (lpop) {
    hipFree(L_vals);
  }
}

void CudaSolver::factor() {
  // For the sake of getting things working, we'll just handle the spd case for now.
  // Later on, we can case on spd and choose Cholesky vs LU accordingly
  if (!spd) {
    return;
  }

  // Boilerplate
  csric02Info_t info;
  hipsparseCreateCsric02Info(&info);

  hipsparseMatDescr_t descr;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
  hipsparseSetMatFillMode(descr, HIPSPARSE_FILL_MODE_LOWER);
  hipsparseSetMatDiagType(descr, HIPSPARSE_DIAG_TYPE_NON_UNIT);

  int bufferSize;
  hipsparseDcsric02_bufferSize(cs_handle, m, nnz, descr, device_vals, device_row_ptr,
                              device_col_idx, info, &bufferSize);

  void *pBuffer;
  hipMalloc(&pBuffer, bufferSize);

  // Analyze
  hipsparseDcsric02_analysis(cs_handle, m, nnz, descr, device_vals,
                            device_row_ptr, device_col_idx, info,
                            HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);

  // Factor, and put the Cholesky factor into L_vals
  hipsparseDcsric02(cs_handle, m, nnz, descr, L_vals,
                   device_row_ptr, device_col_idx, info,
                   HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);

  hipFree(pBuffer);

  hipsparseDestroyMatDescr(descr);

  hipsparseDestroyCsric02Info(info);
  lpop = true;
}

void CudaSolver::solve(double *x) {
  lowerTriangularSolve();

  if (!is_lt) {
    upperTriangularSolve();
  }
  hipMemcpy(x, device_b, m*sizeof(double), hipMemcpyDeviceToHost);
}

void CudaSolver::lowerTriangularSolve() {
  // For the sake of getting things working, we'll just handle the spd case for now.
  // Later on, we can case on spd and choose Cholesky vs LU accordingly
  if (!spd && !is_lt) {
    return;
  }

  // We'll need to access row_ptr and col_idx quite often without modifying them,
  // so store them as global constants
  GlobalConstants params;
  params.row_ptr = device_row_ptr;
  params.col_idx = device_col_idx;
  params.m = m;
  params.nnz = nnz;
  hipMemcpyToSymbol(HIP_SYMBOL(cuConstSolverParams), &params, sizeof(GlobalConstants));

  int *levelInd;
  int *levelPtr;
  int *chainPtr;
  int *rRoot;
  int *wRoot;
  int *cRoot;
  int *scratch;
  int *nRoots;
  int *nCand;
  int *levelIndSize;
  int *levelPtrSize;
  int *chainPtrSize;

  // The maximum number of roots is the number of rows
  hipMalloc(&rRoot, m*sizeof(int));
  hipMalloc(&wRoot, m*sizeof(int));
  hipMalloc(&cRoot, m*sizeof(int));
  hipMalloc(&scratch, m*sizeof(int));
  hipMalloc(&nRoots, sizeof(int));
  hipMalloc(&nCand, sizeof(int));
  hipMalloc(&levelInd, m*sizeof(int));
  hipMalloc(&levelPtr, m*sizeof(int));
  hipMalloc(&chainPtr, m*sizeof(int));
  hipMalloc(&levelIndSize, sizeof(int));
  hipMalloc(&levelPtrSize, sizeof(int));
  hipMalloc(&chainPtrSize, sizeof(int));

  // Sparse binary matrix with the same row pointers and column indices
  // as the LHS. If a row contains all zeros, the corresponding row of the solution
  // has no dependencies, and is therefore a root
  char *depGraph;
  hipMalloc(&depGraph, nnz*sizeof(char));
  hipMemset(depGraph, 1, nnz*sizeof(char));

  // ANALYSIS PHASE

  // Finding roots parallelizes over rows, so we have 1D blocks
  dim3 blockDim(256);
  dim3 gridDim((m + blockDim.x - 1) / blockDim.x);

  // TODO: Naumov only used one kernel for this. What am I doing wrong?
  printf("Finding roots p1\n");
  kernelFindRootsP1<<<gridDim, blockDim>>>(scratch, depGraph);
  hipDeviceSynchronize();
  printf("Scanning\n");
  thrust::inclusive_scan(thrust::device_pointer_cast(scratch),
                         thrust::device_pointer_cast(scratch) + m,
                         thrust::device_pointer_cast(scratch));
  hipDeviceSynchronize();
  printf("Finding roots p2\n");
  kernelFindRootsP2<<<gridDim, blockDim>>>(wRoot, nRoots, scratch);
  hipDeviceSynchronize();

  int nCand_host = 0;

  // Only for debugging
  int *wRoot_host = (int*)malloc(m*sizeof(int));
  int nRoots_host = 0;
  while (true) {
    // TODO: replaced rRoot with wRoot, seems like rRoot unnecessary?
    //kernelAnalyze<<<gridDim, blockDim>>>(wRoot, nRoots, cRoot, nCand, levelInd, levelPtr, chainPtr, levelIndSize, levelPtrSize, chainPtrSize, depGraph);
    //
    hipMemcpy(&nRoots_host, nRoots, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(wRoot_host, wRoot, m*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < nRoots_host; ++i) {
      printf("%d ", wRoot[i]);
    }
    printf("\n");

    hipMemcpy(&nCand_host, nCand, sizeof(int), hipMemcpyDeviceToHost);
    if (nCand_host == 0) {
      break;
    }

    // TODO: again, Naumov did this with one kernel
    kernelFindRootsInCandidatesP1<<<gridDim, blockDim>>>(scratch, cRoot, nCand, depGraph);
    thrust::inclusive_scan(thrust::device_pointer_cast(scratch),
                           thrust::device_pointer_cast(scratch) + m,
                           thrust::device_pointer_cast(scratch));
    kernelFindRootsP2<<<gridDim, blockDim>>>(wRoot, nRoots, scratch);
  }

  free(wRoot_host);

  // SOLVE PHASE
  // TODO: solve phase


  hipFree(rRoot);
  hipFree(wRoot);
  hipFree(cRoot);
  hipFree(scratch);
  hipFree(nRoots);
  hipFree(nCand);
  hipFree(depGraph);
  hipFree(levelInd);
  hipFree(levelPtr);
  hipFree(chainPtr);
  hipFree(levelIndSize);
  hipFree(levelPtrSize);
  hipFree(chainPtrSize);
}

void CudaSolver::upperTriangularSolve() {
  // lol
}
