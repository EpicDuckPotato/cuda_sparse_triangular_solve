#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include "cudaSolver.h"
#include <malloc.h>
#include <sys/time.h>

#define THREADS_PER_BLOCK 256

struct GlobalConstants {
  int *row_ptr;
  int *col_idx;
  int m;
  int nnz;

  // Only for dependency graph
  int *col_ptr;
  int *row_idx;
};

__constant__ GlobalConstants cuConstSolverParams;

// LOWER TRIANGULAR KERNELS. ASSUME DIAGONAL CONTAINS ONES

/*
 * kernelFindRootsL: parallelizes over rows of the dependency
 * graph and indicates roots (for lower triangular)
 * ARGUMENTS
 * roots: roots[i] is populated with 1 if row i is a root, and zero otherwise
 * depGraph: value array for the dependency graph
 */
__global__ void kernelFindRootsL(int *roots, char *depGraph) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < cuConstSolverParams.m) {
    roots[row] = 1;
    for (int i = cuConstSolverParams.row_ptr[row];
         cuConstSolverParams.col_idx[i] < row && i < cuConstSolverParams.row_ptr[row + 1]; ++i) {
      if (depGraph[i]) {
        // Dependency exists
        roots[row] = 0;
        break;
      }
    }
  }
}

/*
 * kernelFindRootsInCandidatesL: parallelizes over rows of the dependency
 * graph and indicates roots, only looking at rows given by cRoot (for lower triangular)
 * ARGUMENTS
 * roots: roots[i] is populated with 1 if row i is a root, and zero otherwise
 * cRoot: 0-1 array indicating candidates
 * nCand: number of candidates
 * depGraph: value array for the dependency graph
 */
__global__ void kernelFindRootsInCandidatesL(int *roots, char *cRoot, char *depGraph) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < cuConstSolverParams.m) {
    if (cRoot[row] == 1) {
      roots[row] = 1;
      for (int i = cuConstSolverParams.row_ptr[row];
           i < cuConstSolverParams.row_ptr[row + 1] &&
           cuConstSolverParams.col_idx[i] < row; ++i) {
        if (depGraph[i]) {
          // Dependency exists
          roots[row] = 0;
          break;
        }
      }
    } else {
      roots[row] = 0;
    }
  }
}

/*
 * kernelFindRootsU: parallelizes over rows of the dependency
 * graph and indicates roots (for upper triangular)
 * ARGUMENTS
 * roots: roots[i] is populated with 1 if row i is a root, and zero otherwise
 * depGraph: value array for the dependency graph
 */
__global__ void kernelFindRootsU(int *roots, char *depGraph) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < cuConstSolverParams.m) {
    int rowStart = cuConstSolverParams.row_ptr[row];
    int rowEnd = cuConstSolverParams.row_ptr[row + 1] - 1;

    roots[row] = 1;
    for (int i = rowEnd; cuConstSolverParams.col_idx[i] > row && i >= rowStart; --i) {
      if (depGraph[i]) {
        // Dependency exists
        roots[row] = 0;
        break;
      }
    }
  }
}

/*
 * kernelFindRootsInCandidatesU: parallelizes over rows of the dependency
 * graph and indicates roots, only looking at rows given by cRoot (for upper triangular)
 * ARGUMENTS
 * roots: roots[i] is populated with 1 if row i is a root, and zero otherwise
 * cRoot: 0-1 array indicating candidates
 * nCand: number of candidates
 * depGraph: value array for the dependency graph
 */
__global__ void kernelFindRootsInCandidatesU(int *roots, char *cRoot, char *depGraph) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < cuConstSolverParams.m) {
    if (cRoot[row] == 1) {
      int rowStart = cuConstSolverParams.row_ptr[row];
      int rowEnd = cuConstSolverParams.row_ptr[row + 1] - 1;

      roots[row] = 1;
      for (int i = rowEnd;
           i >= rowStart && cuConstSolverParams.col_idx[i] > row; --i) {
        if (depGraph[i]) {
          // Dependency exists
          roots[row] = 0;
          break;
        }
      }
    } else {
      roots[row] = 0;
    }
  }
}

/*
 * kernelAnalyze: populates levelInd, levelPtr, and cRoot.
 * chainPtr determines the properties and number of kernels to be launched in the solve phase.
 * ARGUMENTS
 * cRoot: candidates are indicated with a 1. We set the rows of any current roots to 0
 * levelInd: sorted rows belonging to each level. We add rows on this level to the end
 * levelPtr: starting indices (in levelInd) of each level.
 * We add the starting index of the NEXT level to the end. If level == 0, we also
 * make levelPtr[0] = 0
 * nRoots: populated with number of roots
 * rootScan: inclusive scan of the 0-1 array indicating roots
 * rowsDone: how many rows have we already added to levelInd
 * (i.e. at what index in levelInd should we start adding things)?
 * level: what level is this?
 * depGraph: value array for the dependency graph
 */
__global__ void kernelAnalyze(char *cRoot, int *levelInd, int *levelPtr, int *nRoots,
                              int *rootScan, int rowsDone, int level, char *depGraph) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < cuConstSolverParams.m &&
      ((row == 0 && rootScan[row] == 1) ||
       (row > 0 && rootScan[row] == rootScan[row - 1] + 1))) {
    levelInd[rowsDone + rootScan[row] - 1] = row;
    cRoot[row] = 0;

    // Eliminate dependencies
    for (int i = cuConstSolverParams.col_ptr[row];
         i < cuConstSolverParams.col_ptr[row + 1]; ++i) {
      int crow = cuConstSolverParams.row_idx[i];
      for (int j = cuConstSolverParams.row_ptr[crow];
           j < cuConstSolverParams.row_ptr[crow + 1]; ++j) {
        if (cuConstSolverParams.col_idx[j] == row) {
          depGraph[j] = 0;
        }
      }
    }
  }
  if (level == 0) {
    levelPtr[level] = 0;
  }
  levelPtr[level + 1] = rowsDone + rootScan[cuConstSolverParams.m - 1];
  *nRoots = rootScan[cuConstSolverParams.m - 1];
}

/*
 * kernelMultiblockL: processes a single level for lower triangular
 * ARGUMENTS
 * start: start of chain
 * levelInd: sorted rows belonging to each level
 * levelPtr: starting indices (in levelInd) of each level
 * b: b matrix, populated with solution
 * val: L matrix values
 */
__global__ void kernelMultiblockL(int start, int *levelInd, int *levelPtr, double *b, double *val) {
  int startIdx = levelPtr[start];
  int endIdx = levelPtr[start + 1];

  int idx = startIdx + blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < endIdx) {
    // Compute element of solution corresponding to row
    int row = levelInd[idx];
    for (int i = cuConstSolverParams.row_ptr[row];
         i < cuConstSolverParams.row_ptr[row + 1] && cuConstSolverParams.col_idx[i] < row; ++i) {
      b[row] -= val[i] * b[cuConstSolverParams.col_idx[i]];
    }
  }
}

/*
 * kernelSingleblockL: processes a chain for lower triangular
 * ARGUMENTS
 * start: start of chain
 * end: end of chain
 * levelInd: sorted rows belonging to each level
 * levelPtr: starting indices (in levelInd) of each level
 * b: b matrix, populated with solution
 * val: L matrix values
 */
__global__ void kernelSingleblockL(int start, int end, int *levelInd, int *levelPtr, double *b, double *val) {
  int startIdx;
  int endIdx;

  for (int i = start; i < end; ++i) {
    startIdx = levelPtr[i];
    endIdx = levelPtr[i + 1];

    int idx = startIdx + blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < endIdx) {
      // Compute element of solution corresponding to row
      int row = levelInd[idx];
      for (int i = cuConstSolverParams.row_ptr[row];
           i < cuConstSolverParams.row_ptr[row + 1] && cuConstSolverParams.col_idx[i] < row; ++i) {
        b[row] -= val[i] * b[cuConstSolverParams.col_idx[i]];
      }
    }
    __syncthreads();
  }
}

/*
 * kernelMultiblockU: processes a single level for upper triangular
 * ARGUMENTS
 * start: start of chain
 * levelInd: sorted rows belonging to each level
 * levelPtr: starting indices (in levelInd) of each level
 * b: b matrix, populated with solution
 * val: L matrix values
 */
__global__ void kernelMultiblockU(int start, int *levelInd, int *levelPtr, double *b, double *val) {
  int startIdx = levelPtr[start];
  int endIdx = levelPtr[start + 1];

  int idx = startIdx + blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < endIdx) {
    // Compute element of solution corresponding to row
    int row = levelInd[idx];
    int rowStart = cuConstSolverParams.row_ptr[row];
    int rowEnd = cuConstSolverParams.row_ptr[row + 1] - 1;

    int i = rowEnd;
    for (; i >= rowStart && cuConstSolverParams.col_idx[i] > row; --i) {
      b[row] -= val[i] * b[cuConstSolverParams.col_idx[i]];
    }
    b[row] /= val[i];
  }
}

/*
 * kernelSingleblockU: processes a chain for upper triangular
 * ARGUMENTS
 * start: start of chain
 * end: end of chain
 * levelInd: sorted rows belonging to each level
 * levelPtr: starting indices (in levelInd) of each level
 * b: b matrix, populated with solution
 * val: L matrix values
 */
__global__ void kernelSingleblockU(int start, int end, int *levelInd, int *levelPtr, double *b, double *val) {
  int startIdx;
  int endIdx;

  for (int i = start; i < end; ++i) {
    startIdx = levelPtr[i];
    endIdx = levelPtr[i + 1];

    int idx = startIdx + blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < endIdx) {
      // Compute element of solution corresponding to row
      int row = levelInd[idx];
      int rowStart = cuConstSolverParams.row_ptr[row];
      int rowEnd = cuConstSolverParams.row_ptr[row + 1] - 1;

      int i = rowEnd;
      for (; i >= rowStart && cuConstSolverParams.col_idx[i] > row; --i) {
        b[row] -= val[i] * b[cuConstSolverParams.col_idx[i]];
      }
      b[row] /= val[i];
    }
    __syncthreads();
  }
}

CudaSolver::CudaSolver(int *row_idx, int *col_idx, double *vals, int m, int nnz, double *b) : m(m), nnz(nnz) {
  hipsparseCreate(&cs_handle);

  hipMalloc(&device_row_ptr, sizeof(int)*(m + 1));
  hipMalloc(&device_col_ptr, sizeof(int)*(m + 1));
  hipMalloc(&device_row_idx, sizeof(int)*nnz);
  hipMalloc(&device_col_idx, sizeof(double)*nnz);
  hipMalloc(&device_vals, sizeof(double)*nnz);
  hipMemcpy(device_row_idx, row_idx, sizeof(int)*nnz, hipMemcpyHostToDevice);
  hipMemcpy(device_col_idx, col_idx, sizeof(int)*nnz, hipMemcpyHostToDevice);
  hipMemcpy(device_vals, vals, sizeof(double)*nnz, hipMemcpyHostToDevice);

  thrust::device_vector<int> device_row_idx2(device_row_idx, device_row_idx + nnz);
  thrust::device_vector<int> device_row_idx3(device_row_idx3);

  // Convert COO to CSC (for efficient iteration through dependency graph)
  hipsparseXcoo2csr(cs_handle, device_col_idx, nnz, m, device_col_ptr,
                   HIPSPARSE_INDEX_BASE_ZERO);

  // We assume col_idx is sorted because that's how the matrices from matrix market are.
  thrust::stable_sort_by_key(thrust::device,
                             device_row_idx2.begin(),
                             device_row_idx2.end(),
                             thrust::device_pointer_cast(device_col_idx));

  thrust::stable_sort_by_key(thrust::device,
                             device_row_idx3.begin(),
                             device_row_idx3.end(),
                             thrust::device_pointer_cast(device_vals));

  // Convert COO to CSR
  hipsparseXcoo2csr(cs_handle, thrust::raw_pointer_cast(device_row_idx2.data()), nnz, m, device_row_ptr,
                   HIPSPARSE_INDEX_BASE_ZERO);

  hipMalloc(&device_b, sizeof(double)*m);
  hipMemcpy(device_b, b, sizeof(double)*m, hipMemcpyHostToDevice);
}

CudaSolver::~CudaSolver() {
  hipsparseDestroy(cs_handle);
  hipFree(device_row_ptr);
  hipFree(device_col_idx);
  hipFree(device_col_ptr);
  hipFree(device_row_idx);
  hipFree(device_vals);
  hipFree(device_b);
}

void CudaSolver::factor() {
  // Boilerplate
  csrilu02Info_t info = 0;
  hipsparseCreateCsrilu02Info(&info);

  hipsparseMatDescr_t descr = 0;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);

  int bufferSize;
  hipsparseDcsrilu02_bufferSize(cs_handle, m, nnz, descr, device_vals, device_row_ptr,
                               device_col_idx, info, &bufferSize);

  void *pBuffer;
  hipMalloc(&pBuffer, bufferSize);

  // Analyze
  hipsparseDcsrilu02_analysis(cs_handle, m, nnz, descr, device_vals,
                             device_row_ptr, device_col_idx, info,
                             HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);

  // Factor, and put the factors into device_vals
  hipsparseDcsrilu02(cs_handle, m, nnz, descr, device_vals,
                    device_row_ptr, device_col_idx, info,
                    HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);

  hipFree(pBuffer);

  hipsparseDestroyMatDescr(descr);

  hipsparseDestroyCsrilu02Info(info);
}

void CudaSolver::get_factors(int *row_ptr_L, int *col_idx_L, double *vals_L,
                             int *row_ptr_U, int *col_idx_U, double *vals_U) {
  int *full_row_ptr = (int*)malloc(sizeof(int)*(m + 1));
  int *col_idx = (int*)malloc(sizeof(int)*nnz);
  double *vals = (double*)malloc(sizeof(double)*nnz);
  hipMemcpy(full_row_ptr, device_row_ptr, (m + 1)*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(col_idx, device_col_idx, nnz*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(vals, device_vals, nnz*sizeof(double), hipMemcpyDeviceToHost);

  int iL = 0;
  int iU = 0;
  int row = 0;

  for (row = 0; row < m; ++row) {
    // Lower
    row_ptr_L[row] = iL;
    int j = full_row_ptr[row];
    for (; col_idx[j] < row && j < full_row_ptr[row + 1]; ++j) {
      col_idx_L[iL] = col_idx[j];
      vals_L[iL] = vals[j];
      ++iL;
    }

    col_idx_L[iL] = row;
    vals_L[iL] = 1;
    ++iL;

    // Upper
    row_ptr_U[row] = iU;
    for (; j < full_row_ptr[row + 1]; ++j) {
      col_idx_U[iU] = col_idx[j];
      vals_U[iU] = vals[j];
      ++iU;
    }
  }
  row_ptr_L[row] = iL;
  row_ptr_U[row] = iU;

  free(full_row_ptr);
  free(col_idx);
  free(vals);
}

void CudaSolver::solve(double *x) {
  printf("lower\n");
  triangularSolve(true);
  printf("upper\n");
  triangularSolve(false);
  hipMemcpy(x, device_b, m*sizeof(double), hipMemcpyDeviceToHost);
}

void CudaSolver::triangularSolve(bool isLower) {
  // We'll need to access row_ptr and col_idx quite often without modifying them,
  // so store them as global constants
  GlobalConstants params;
  params.row_ptr = device_row_ptr;
  params.col_idx = device_col_idx;
  params.m = m;
  params.nnz = nnz;
  params.col_ptr = device_col_ptr;
  params.row_idx = device_row_idx;
  hipMemcpyToSymbol(HIP_SYMBOL(cuConstSolverParams), &params, sizeof(GlobalConstants));

  int *levelInd;
  hipMalloc(&levelInd, m*sizeof(int));

  int *levelPtr;
  hipMalloc(&levelPtr, (m + 1)*sizeof(int)); // Worst-case scenario, each level contains a single row, and we need a pointer to the end, so m + 1

  // We can have max THREADS_PER_BLOCK rows in a chain, so there can be max
  // (m + THREADS_PER_BLOCK)/THREADS_PER_BLOCK chains (accounting for integer division)
  int *chainPtr = (int*)malloc(sizeof(int)*(m + THREADS_PER_BLOCK)/THREADS_PER_BLOCK + 1);

  int *rRoot;
  hipMalloc(&rRoot, m*sizeof(int)); // The maximum number of roots is the number of rows
  hipMemset(rRoot, 0, m*sizeof(int));

  char *cRoot;
  hipMalloc(&cRoot, m*sizeof(char));
  hipMemset(cRoot, 1, m*sizeof(char)); // Everything's a candidate at first

  int *nRoots;
  hipMalloc(&nRoots, sizeof(int));

  // Sparse binary matrix with the same row pointers and column indices
  // as the LHS. If a row contains all zeros, the corresponding row of the solution
  // has no dependencies, and is therefore a root
  char *depGraph;
  hipMalloc(&depGraph, nnz*sizeof(char));
  hipMemset(depGraph, 1, nnz*sizeof(char));

  // ANALYSIS PHASE

  // Finding roots parallelizes over rows, so we have 1D blocks
  dim3 blockDim(THREADS_PER_BLOCK);
  dim3 gridDim((m + blockDim.x - 1) / blockDim.x);


  // Get 0-1 array of roots
  if (isLower) {
    kernelFindRootsL<<<gridDim, blockDim>>>(rRoot, depGraph);
  } else {
    kernelFindRootsU<<<gridDim, blockDim>>>(rRoot, depGraph);
  }
  hipDeviceSynchronize();

  thrust::inclusive_scan(thrust::device_pointer_cast(rRoot),
                         thrust::device_pointer_cast(rRoot) + m,
                         thrust::device_pointer_cast(rRoot));

  int nRoots_host = 0;
  int level = 0;
  int rowsDone = 0;
  int rowsInChain = 0;
  int chainIdx = 0;
  chainPtr[chainIdx] = level;

  //struct timeval t1, t2;

  // Upon exiting, chainIdx contains the number of chains
  while (true) {
    //gettimeofday(&t1, 0);
    kernelAnalyze<<<gridDim, blockDim>>>(cRoot, levelInd, levelPtr,
                                         nRoots, rRoot, rowsDone, level, depGraph);
    hipDeviceSynchronize();

    /*
    gettimeofday(&t2, 0);
    double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    printf("Time to analyze roots:  %3.1f ms \n", time);
    */

    hipMemcpy(&nRoots_host, nRoots, sizeof(int), hipMemcpyDeviceToHost);
    if (nRoots_host == 0) {
      chainPtr[++chainIdx] = level;
      // Now the last element of chainPtr contains the number of levels
      break;
    }

    ++level;

    if (rowsInChain + nRoots_host > THREADS_PER_BLOCK) {
      // Adding this new level of roots to the current chain
      // would cause us to overflow the current chain. Add a new
      // chain starting at this level
      chainPtr[++chainIdx] = level;
      rowsInChain = 0;
    }

    rowsInChain += nRoots_host;
    rowsDone += nRoots_host;

    // Get 0-1 array of roots
    if (isLower) {
      kernelFindRootsInCandidatesL<<<gridDim, blockDim>>>(rRoot, cRoot, depGraph);
    } else {
      kernelFindRootsInCandidatesU<<<gridDim, blockDim>>>(rRoot, cRoot, depGraph);
    }
    hipDeviceSynchronize();

    thrust::inclusive_scan(thrust::device_pointer_cast(rRoot),
                           thrust::device_pointer_cast(rRoot) + m,
                           thrust::device_pointer_cast(rRoot));
  }

  // SOLVE PHASE

  int start;
  int end;

  dim3 gridDimOneBlock(1);
  // Iterate over chains
  for (int i = 0; i < chainIdx; ++i) {
    start = chainPtr[i];
    end = chainPtr[i+1];

    // Process a chain
    if (end - start > 1) {
      if (isLower) {
        kernelSingleblockL<<<gridDimOneBlock, blockDim>>>(start, end, levelInd, levelPtr, device_b, device_vals);
      } else {
        kernelSingleblockU<<<gridDimOneBlock, blockDim>>>(start, end, levelInd, levelPtr, device_b, device_vals);
      }
    }
    // Process a single level
    else {
      if (isLower) {
        kernelMultiblockL<<<gridDim, blockDim>>>(start, levelInd, levelPtr, device_b, device_vals);
      } else {
        kernelMultiblockU<<<gridDim, blockDim>>>(start, levelInd, levelPtr, device_b, device_vals);
      }
    }
  }

  printf("Freeing levelInd\n");
  hipFree(levelInd);
  printf("Freeing levelPtr\n");
  hipFree(levelPtr);
  printf("Freeing chainPtr\n");
  free(chainPtr);
  printf("Freeing rRoot\n");
  hipFree(rRoot);
  printf("Freeing cRoot\n");
  hipFree(cRoot);
  printf("Freeing nRoots\n");
  hipFree(nRoots);
  printf("Freeing depGraph\n");
  hipFree(depGraph);
}
